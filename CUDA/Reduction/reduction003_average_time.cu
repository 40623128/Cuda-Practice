
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
const int N               = (1 <<30);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;
//�p�⥭���ɶ������榸��
const int times_of_average = 10;

//�ĤT��Reduction
__global__ void kernel3(float* arr, float* out, int N){
    __shared__ float s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; 
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();

    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(tid < s && i + s < N){
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;
    for(int j=0;j<times_of_average;j++){
        //�D�����s���t
        hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
        hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
        //��d���s���t
        hipMalloc(&a_device, N * sizeof(float));
        hipMalloc(&r_device, blocksPerGrid * sizeof(float));
        //�D�إͦ�
        for(int i=0;i<N;i++){
            a_host[i] = 1;
        }
        for(int i=0;i<blocksPerGrid;i++){
            r_host[i] = 0.0;
        }
        //�w�q��d�y
        hipStream_t stream;
        //�Ыجy
        hipStreamCreate(&stream);

        //�O����]�w(���B)
        hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

        //�w�q�P�Ыض}�l�M����ƥ�(Event)
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        //�����}�l�ƥ�(Event)
        hipEventRecord(start, 0);
        //�B��Kernel1�i��B��
        for(int i=0;i<iters;i++){
            kernel3<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
        }
        //��������ƥ�(Event)
        hipEventRecord(stop, 0);
        //���ݰ���ƥ�(Event)����
        hipEventSynchronize(stop);
        float elapsedTime;
        //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
        hipEventElapsedTime(&elapsedTime, start, stop);
        cout << "GPU Elapse time "<<j<<" : " << elapsedTime / iters << " ms" << endl;
        total_time = total_time + (elapsedTime / iters);
        //�ƥ󲾰�
        hipEventDestroy(start);
        hipEventDestroy(stop);
        //�D���P�]�ƶ��O����ƻs
        hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
        //varifyOutput(r_host, a_host, N);
        //����O����
        hipFree(r_device);
        hipFree(a_device);
        hipHostFree(r_host);
        hipHostFree(a_host);
        //return 0;
}

cout << "GPU Elapse average time for " << times_of_average <<" times:"<< total_time/times_of_average << " ms" << endl;
return 0;
}
