
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
//�ۥ[�������Ӽ�(2^30-3)
const int N               = (1 <<28);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;
const int kernal_number = 7;

//kernel1
__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //�C�ӽu�{Ū���@�Ӥ���
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

//kernel2
__global__ void kernel2(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    // s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        int index = tid * 2 * s;
        if((index + s) < blockDim.x && (blockIdx.x * blockDim.x + index + s) < N){
            s_data[index] += s_data[index + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

//kernel3
__global__ void kernel3(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();

    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(tid < s && i + s < N){
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

//kernel4
__global__ void kernel4(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2); // 3���Ĥ@?���N�A���@�b��?�{�Oidle���A?�b��@?block���j�p?�p�@�b
    if(i < N){
        s_data[tid] = arr[i] + arr[i + blockDim.x];  // ???���?���Ĥ@?���N�A�Z���N?����?
    }
    __syncthreads();

    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(tid < s && i + s < N){
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

//kernel5
__device__ void warpRecude(volatile double* s_data, int tid){ // volatile ??�r�ܭ��n�A�O?s_data?��?��?�s?�����X�A?��??��gpu?�s
    s_data[tid] += s_data[tid + 32];
    s_data[tid] += s_data[tid + 16];
    s_data[tid] += s_data[tid + 8];
    s_data[tid] += s_data[tid + 4];
    s_data[tid] += s_data[tid + 2];
    s_data[tid] += s_data[tid + 1];
}
__global__ void kernel5(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2); // 3���Ĥ@?���N�A���@�b��?�{�Oidle���A?�b��@?block���j�p?�p�@�b
    if(i < N){
        s_data[tid] = arr[i] + arr[i + blockDim.x];  // ???���?���Ĥ@?���N�A�Z���N?����?
    }else{
        s_data[tid] = 0;
    }
    __syncthreads();

    for(int s = blockDim.x/2; s > 32; s>>=1){
        if(tid < s && i + s < N){
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid < 32){
        warpRecude(s_data, tid);
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

//kernel6
template<unsigned int blockSize>
__device__ void warpRecude06(volatile double* s_data, int tid){ // volatile ??�r�ܭ��n�A�O?s_data?��?��?�s?�����X�A?��??��gpu?�s
    if(blockSize >= 64) s_data[tid] += s_data[tid + 32];   // if �O����blockSize�p�_64�A��pblockSize?16�A���\?������U��
    if(blockSize >= 32) s_data[tid] += s_data[tid + 16];
    if(blockSize >= 16) s_data[tid] += s_data[tid + 8];
    if(blockSize >= 8) s_data[tid] += s_data[tid + 4];
    if(blockSize >= 4) s_data[tid] += s_data[tid + 2];
    if(blockSize >= 2) s_data[tid] += s_data[tid + 1];
}
template<unsigned int blockSize>
__global__ void reduce06(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2); // 3���Ĥ@?���N�A���@�b��?�{�Oidle���A?�b��@?block���j�p?�p�@�b
    if(i < N){
        s_data[tid] = arr[i] + arr[i + blockDim.x];  // ???���?���Ĥ@?���N�A�Z���N?����?
    }else{
        s_data[tid] = 0;
    }
    __syncthreads();

    if(blockSize >= 1024){
        if(tid < 512){
            s_data[tid] += s_data[tid+512];
        }
        __syncthreads();
    }
    if(blockSize >= 512){
        if(tid < 256){
            s_data[tid] += s_data[tid+256];
        }
        __syncthreads();
    }
    if(blockSize >= 256){
        if(tid < 128){
            s_data[tid] += s_data[tid+128];
        }
        __syncthreads();
    }
    if(blockSize >= 128){
        if(tid < 64){
            s_data[tid] += s_data[tid+64];
        }
        __syncthreads();
    }

    if(tid < 32){
        warpRecude06<blockSize>(s_data, tid);
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}
void kernel6(double* arr, double* out, int N, hipStream_t &stream){   // �i?�Ҧ����`?�A�h���`?
    switch(threadsPerBlock){
        case 1024:
            reduce06<1024><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 512:
            reduce06<512><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 256:
            reduce06<256><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 128:
            reduce06<128><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 64:
            reduce06<64><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 32:
            reduce06<32><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 16:
            reduce06<16><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 8:
            reduce06<8><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 4:
            reduce06<4><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 2:
            reduce06<2><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 1:
            reduce06<1><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
    }
}


//kernel7
template<unsigned int blockSize>
__device__ void warpRecude07(volatile double* s_data, int tid){ // volatile �ܭ��n�A�O��s_data�q���������q�椸���X�A�o�̫�gpu���s
    if(blockSize >= 64) s_data[tid] += s_data[tid + 32];   // if �O����blockSize�p��64�A��pblockSize��16�A���|������
    if(blockSize >= 32) s_data[tid] += s_data[tid + 16];
    if(blockSize >= 16) s_data[tid] += s_data[tid + 8];
    if(blockSize >= 8) s_data[tid] += s_data[tid + 4];
    if(blockSize >= 4) s_data[tid] += s_data[tid + 2];
    if(blockSize >= 2) s_data[tid] += s_data[tid + 1];
}
template<unsigned int blockSize>
__global__ void reduce07(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2); // 3���Ĥ@�����N�A���@�b���u�{�O���m���A�{�b��@��block���j�p�Y�p�@�b
    unsigned int gridSize = blockSize*2*gridDim.x;
    s_data[tid] = 0;

    while (i<N){
        s_data[tid] += arr[i] + arr[i+blockSize];
        i += gridSize;
    }
    __syncthreads();

    if(blockSize >= 1024){
        if(tid < 512){
            s_data[tid] += s_data[tid+512];
        }
        __syncthreads();
    }
    if(blockSize >= 512){
        if(tid < 256){
            s_data[tid] += s_data[tid+256];
        }
        __syncthreads();
    }
    if(blockSize >= 256){
        if(tid < 128){
            s_data[tid] += s_data[tid+128];
        }
        __syncthreads();
    }
    if(blockSize >= 128){
        if(tid < 64){
            s_data[tid] += s_data[tid+64];
        }
        __syncthreads();
    }

    if(tid < 32){
        warpRecude07<blockSize>(s_data, tid);
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}
void kernel7(double* arr, double* out, int N, hipStream_t &stream){   // �i�}�Ҧ��`���A�h���`��
    switch(threadsPerBlock){
        case 1024:
            reduce07<1024><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 512:
            reduce07<512><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 256:
            reduce07<256><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 128:
            reduce07<128><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 64:
            reduce07<64><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 32:
            reduce07<32><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 16:
            reduce07<16><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 8:
            reduce07<8><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 4:
            reduce07<4><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 2:
            reduce07<2><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 1:
            reduce07<1><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
    }
}


int main(){
    double* a_host, *r_host;
    double* a_device, *r_device;
    float total_time = 0.0;
    for(int k=0; k<kernal_number; k++){
            //�D�����s���t
            hipHostMalloc(&a_host, N * sizeof(double), hipHostMallocDefault);
            hipHostMalloc(&r_host, blocksPerGrid * sizeof(double), hipHostMallocDefault);
            //��d���s���t
            hipMalloc(&a_device, N * sizeof(double));
            hipMalloc(&r_device, blocksPerGrid * sizeof(double));
            //�D�إͦ�
            for(int i=0;i<N;i++){
                a_host[i] = 1;
            }
            for(int i=0;i<blocksPerGrid;i++){
                r_host[i] = 0.0;
            }
            //�w�q��d�y
            hipStream_t stream;
            //�Ыجy
            hipStreamCreate(&stream);

            //�O����]�w(���B)
            hipMemcpyAsync(a_device, a_host, N * sizeof(double), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(double), hipMemcpyHostToDevice, stream);

            //�w�q�P�Ыض}�l�M����ƥ�(Event)
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            //�����}�l�ƥ�(Event)
            hipEventRecord(start, 0);
            //�B��Kernel1�i��B��
            if (k == 0){
            for(int i=0;i<iters;i++){
                kernel1<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
            }
            }
            if (k == 1){
            for(int i=0;i<iters;i++){
                kernel2<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
            }
            }
            if (k == 2){
            for(int i=0;i<iters;i++){
                kernel3<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
            }
            }
            if (k == 3){
            for(int i=0;i<iters;i++){
                kernel4<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
            }
            }
            if (k == 4){
            for(int i=0;i<iters;i++){
                kernel5<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
            }
            }
            if (k == 5){
            for(int i=0;i<iters;i++){
                kernel6(a_device, r_device, N, stream);
            }
            }
            if (k == 6){
            for(int i=0;i<iters;i++){
                kernel7(a_device, r_device, N, stream);
            }
            }
            //��������ƥ�(Event)
            hipEventRecord(stop, 0);
            //���ݰ���ƥ�(Event)����
            hipEventSynchronize(stop);
            float elapsedTime;
            //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
            hipEventElapsedTime(&elapsedTime, start, stop);
            total_time = total_time + (elapsedTime / iters);
            //�ƥ󲾰�
            hipEventDestroy(start);
            hipEventDestroy(stop);
            //�D���P�]�ƶ��O����ƻs
            hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
            //varifyOutput(r_host, a_host, N);
            //����O����
            hipFree(r_device);
            hipFree(a_device);
            hipHostFree(r_host);
            hipHostFree(a_host);
            cout << "The Kernal" << k+1 <<" times:"<< endl;
            cout << "GPU Elapse time for The Kernal " << k+1 <<" :"<< total_time << " ms" << endl;
            total_time = 0.0 ;
    }
return 0;
}
