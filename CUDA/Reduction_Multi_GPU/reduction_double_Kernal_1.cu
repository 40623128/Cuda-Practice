
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#include <stdio.h>
using namespace std;

//const int num_gpus = 2;
//經測試於RTX3070最佳值為128，接著為256。
const int threadsPerBlock = 128;
//相加之元素個數(2^30-3)
const int Total_N               = (1 <<10 );
const int iters           = 1;
//const int kernal_number = 7;
//kernel1


__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}


int main(){
    int num_gpus;
    int device;
    hipError_t ret;
    hipGetDeviceCount(&num_gpus);
    ret = hipGetDevice(&device);
    printf("num_gpus = %d\n",num_gpus);
    printf("hipGetDevice = %d\n",ret);

    const int N = Total_N/num_gpus;
    const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
    float total_time[num_gpus];
    double* a_host[num_gpus], *r_host[num_gpus];
    double* a_device[num_gpus], *r_device[num_gpus];
    for(int i = 0; i < num_gpus; i++){
        //主機內存分配
        hipHostMalloc(&a_host[i], N * sizeof(double), hipHostMallocDefault);
        hipHostMalloc(&r_host[i], blocksPerGrid * sizeof(double), hipHostMallocDefault);
        //顯卡內存分配
        hipMalloc(&a_device[i], N * sizeof(double));
        hipMalloc(&r_device[i], blocksPerGrid * sizeof(double));
    }
    cout << "Memory Allocation Completed" << endl;


    //題目生成
    cout << "Generating list" << endl;
    for(int i = 0; i < num_gpus; i++){
        for(int j=0;j<N;j++){
            a_host[i][j] = 1.0;
        }
        cout << "a_host "<< i <<" Generating Completed" << endl;
        for(int j=0;j<blocksPerGrid;j++){
            r_host[i][j] = 0.0;
        }
        cout << "r_host "<< i <<" Generating Completed" << endl;
    }

    //定義顯卡流
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipStream_t stream0;
        hipStreamCreate(&stream0);
    }
    cout << "GPU Stream Define Completed" << endl;

    //記憶體設定(異步)
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        ret = hipGetDevice(&device);
        printf("Mem hipGetDevice = %d\n",ret);
        hipMemcpyAsync(a_device[i], a_host[i], N * sizeof(double),
                                       hipMemcpyHostToDevice);
        hipMemcpyAsync(r_device[i], r_host[i], blocksPerGrid * sizeof(double),
                                       hipMemcpyHostToDevice);
        printf("Mem ERROR GPU %d = %s\n",i,hipGetErrorString(hipGetLastError()));
    }
    cout << "Memory asynchronous Completed" << endl;

    //定義開始和停止事件(Event)
    hipEvent_t start_events[num_gpus];
    hipEvent_t stop_events[num_gpus];

    //創建開始和停止事件(Event)
    for(int i = 0; i < num_gpus; i++){
     hipSetDevice(i);
     hipEventCreate(&start_events[i]);
     hipEventCreate(&stop_events[i]);
    }
    cout << "Create Start & Stop Event Completed" << endl;
    cout << "Start Calculation" << endl;
    for(int j=0;j<iters;j++){
        for(int i = 0; i < num_gpus; i++){
            hipSetDevice(i);
            hipEventRecord(start_events[i]);
            //運用Kernel1進行運算
            kernel1<<<blocksPerGrid, threadsPerBlock, 0>>>(a_device[i], r_device[i], N);
            //紀錄停止事件(Event)
            hipEventRecord(stop_events[i]);
            hipDeviceSynchronize();
            hipEventSynchronize(stop_events[i]);
            printf("GPU %d ERROR = %s\n",i,hipGetErrorString(hipGetLastError()));
        }
    }
    cout << "Calculation Completed" << endl;


    cout << "Calculation time" << endl;
    float elapsedTime[num_gpus];
    //計算開始事件至暫停事件所經時間
    for(int i = 0; i < num_gpus; i++){
        hipEventElapsedTime(&elapsedTime[i], start_events[i], stop_events[i]);
        total_time[i] = total_time[i] + (elapsedTime[i] / iters);
        //cout << "total_time "<< i << "= " << total_time[i] << endl;
        //cout << "elapsedTime "<< i << "= " << elapsedTime[i] << endl;
    }

    for(int i = 0; i < num_gpus; i++){
        if (i ==0){
            total_time[i] = total_time[i];
        }
        else{
        total_time[i] = total_time[i-1] + total_time[i];
        }
        }


    cout << "Event Destroy" << endl;
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipEventDestroy(start_events[i]);
        hipEventDestroy(stop_events[i]);
    }

    cout << "Share Memory form Device to Host" << endl;
    //資料由顯卡記憶體傳輸至主機記憶體
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipMemcpy(r_host[i], r_device[i], blocksPerGrid * sizeof(double),
                                  hipMemcpyDeviceToHost);
    }

    cout << "Free Memory" << endl;


    for(int i = 0; i < num_gpus; i++){
        cout << "GPU "<< i <<" Elapse time for The Kernal 1 :"<< total_time[i] << " ms" << endl;
        total_time[i] = 0.0 ;
        elapsedTime[i] = 0.0 ;
    }

    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        for(int j = 0; j < blocksPerGrid; j++){
            if (i == 0 && j == 0){
            r_host[0][0] = r_host[i][j];
            }
            else if (r_host[i][j] != 0){
            r_host[0][0] = r_host[0][0] + r_host[i][j];
            }
            printf("r_host[%d][%d] = %f\n", i, j, r_host[i][j]);
            printf("Ans [%d][%d] = %f\n", i, j, r_host[0][0]);
        }
    }
    printf("r_host[0][0] = %f\n",r_host[0][0]);

    //釋放記憶體
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        //cudaFree(r_device[i]);
        hipHostFree(r_host[i]);
        //cudaFree(a_device[i]);
        hipHostFree(a_host[i]);
    }
    return 0;
}
