
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;

//const int num_GPUs = 2;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
//�ۥ[�������Ӽ�(2^30-3)
const int N               = (1 <<28);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;
//const int kernal_number = 7;
//kernel1
__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //�C�ӽu�{Ū���@�Ӥ���
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    double* a_A_host, *r_A_host,*a_B_host,*r_B_host;
    double* a_A_device, *r_A_device, *a_B_device, *r_B_device;
    float total_time = 0.0;
    //�D�����s���t
    hipHostMalloc(&a_A_host, N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&a_B_host, N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&r_A_host, blocksPerGrid * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&r_B_host, blocksPerGrid * sizeof(double), hipHostMallocDefault);
    //��d���s���t
    hipMalloc(&a_A_device, N * sizeof(double));
    hipMalloc(&r_A_device, blocksPerGrid * sizeof(double));
    hipMalloc(&a_B_device, N * sizeof(double));
    hipMalloc(&r_B_device, blocksPerGrid * sizeof(double));
    //�D�إͦ�
    for(int i=0;i<N/2;i++){
        a_A_host[i] = 1;
        a_B_host[i] = 1;
    }
    for(int i=0;i<blocksPerGrid;i++){
        r_A_host[i] = 0.0;
        r_B_host[i] = 0.0;
    }
    //�w�q��d�y
    hipStream_t streamA, streamB;
    //�Ыجy
    hipSetDevice(0);
    hipStreamCreate(&streamA);
    hipSetDevice(1);
    hipStreamCreate(&streamB);

    //�O����]�w(���B)
    hipMemcpyAsync(a_A_device, a_A_host, N * sizeof(double), hipMemcpyHostToDevice, streamA);
    hipMemcpyAsync(r_A_device, r_A_host, blocksPerGrid * sizeof(double), hipMemcpyHostToDevice, streamA);
    hipMemcpyAsync(a_B_device, a_B_host, N * sizeof(double), hipMemcpyHostToDevice, streamB);
    hipMemcpyAsync(r_B_device, r_B_host, blocksPerGrid * sizeof(double), hipMemcpyHostToDevice, streamB);
    //�w�q�P�Ыض}�l�M����ƥ�(Event)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //�����}�l�ƥ�(Event)
    hipEventRecord(start, 0);
    //�B��Kernel1�i��B��

    for(int i=0;i<iters;i++){
        kernel1<<<blocksPerGrid, threadsPerBlock, 0, streamA>>>(a_A_device, r_A_device, N/2);
        kernel1<<<blocksPerGrid, threadsPerBlock, 0, streamB>>>(a_B_device, r_B_device, N/2);
    }

    //��������ƥ�(Event)
    hipEventRecord(stop, 0);
    //���ݰ���ƥ�(Event)����
    hipEventSynchronize(stop);
    float elapsedTime;
    //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
    hipEventElapsedTime(&elapsedTime, start, stop);
    total_time = total_time + (elapsedTime / iters);
    //�ƥ󲾰�
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //�D���P�]�ƶ��O����ƻs
    hipMemcpy(r_A_host, r_A_device, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(r_B_host, r_B_device, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
    //varifyOutput(r_host, a_host, N);
    //����O����
    hipFree(r_A_device);
    hipFree(a_A_device);
    hipFree(r_B_device);
    hipFree(a_B_device);
    hipHostFree(r_A_host);
    hipHostFree(a_A_host);
    hipHostFree(r_B_host);
    hipHostFree(a_B_host);
    cout << "GPU Elapse time for The Kernal 1" <<" :"<< total_time << " ms" << endl;
    total_time = 0.0 ;
    return 0;
}
