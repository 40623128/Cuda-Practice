
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#include <stdio.h>
using namespace std;

//const int num_gpus = 2;
//經測試於RTX3070最佳值為128，接著為256。
const int threadsPerBlock = 128;
//相加之元素個數(2^30-3)
const int Total_N               = (1 <<20 );
const int iters           = 1;
//const int kernal_number = 7;
//kernel1


__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}


int main(){
    int num_gpus;
    int device;
    hipError_t err;
    hipGetDeviceCount(&num_gpus);
    printf("num_gpus = %d\n",num_gpus);
    err = hipGetDevice(&device);
    printf("hipGetDevice = %d\n",err);

    const int N = Total_N/num_gpus;
    const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
    float total_time[num_gpus];
    double* a_host[num_gpus], *r_host[num_gpus];
    double* a_device[num_gpus], *r_device[num_gpus];
    for(int i = 0; i < num_gpus; i++){
        //主機內存分配
        hipHostMalloc(&a_host[i], N * sizeof(double), hipHostMallocDefault);
        hipHostMalloc(&r_host[i], blocksPerGrid * sizeof(double), hipHostMallocDefault);
        //顯卡內存分配
    }
    cout << "Memory Allocation Completed" << endl;


    //題目生成
    cout << "Generating list" << endl;
    for(int i = 0; i < num_gpus; i++){
        for(int j=0;j<N;j++){
            a_host[i][j] = 1.0+i*N+j;
        }
        cout << "a_host "<< i <<" Generating Completed" << endl;
        for(int j=0;j<blocksPerGrid;j++){
            r_host[i][j] = 0.0;
        }
        cout << "r_host "<< i <<" Generating Completed" << endl;
    }


    //定義開始和停止事件(Event)
    hipStream_t stream[num_gpus];
    hipEvent_t start_events[num_gpus];
    hipEvent_t stop_events[num_gpus];
    float elapsedTime[num_gpus];

    //定義顯卡流
    for(int i = 0; i < num_gpus; i++){
        err = hipSetDevice(i);
        printf("GPU %d Set Device = %d\n",i,err);
        err = hipGetDevice(&device);
        printf("GPU %d Get Device = %d\n",i,err);
        //定義顯卡流
        hipStreamCreate(&stream[i]);
        printf("GPU %d Stream Define Completed\n",i);

        hipMalloc(&a_device[i], N * sizeof(double));
        hipMalloc(&r_device[i], blocksPerGrid * sizeof(double));



        //記憶體設定(異步)
        hipMemcpyAsync(a_device[i], a_host[i], N * sizeof(double),
                        hipMemcpyHostToDevice,stream[i]);
        hipMemcpyAsync(r_device[i], r_host[i], blocksPerGrid * sizeof(double),
                        hipMemcpyHostToDevice,stream[i]);
        printf("Mem ERROR GPU %d = %s\n",i,hipGetErrorString(hipGetLastError()));
        printf("Memory asynchronous Completed\n");

        //創建開始和停止事件(Event)
        hipEventCreate(&start_events[i]);
        hipEventCreate(&stop_events[i]);
        printf("Create Start & Stop Event Completed\n");

        printf("Start Calculation\n");
        hipEventRecord(start_events[i],stream[i]);
        //運用Kernel1進行運算
        kernel1<<<blocksPerGrid, threadsPerBlock, 0,stream[i]>>>(a_device[i], r_device[i], N);
        //紀錄停止事件(Event)
        hipEventRecord(stop_events[i],stream[i]);
        hipDeviceSynchronize();
        hipEventSynchronize(stop_events[i]);
        printf("GPU %d ERROR = %s\n",i,hipGetErrorString(hipGetLastError()));
        printf("GPU %d Calculation Completed\n",i);

        //計算開始事件至暫停事件所經時間
        printf("GPU %d Calculation time\n",i);
        hipEventElapsedTime(&elapsedTime[i], start_events[i], stop_events[i]);
        total_time[i] = total_time[i] + (elapsedTime[i] / iters);
        //cout << "total_time "<< i << "= " << total_time[i] << endl;
        //cout << "elapsedTime "<< i << "= " << elapsedTime[i] << endl;
        if (i ==0){
            total_time[i] = total_time[i];
        }
        else{
            total_time[i] = total_time[i-1] + total_time[i];
        }

        cout << "Event Destroy" << endl;
        hipEventDestroy(start_events[i]);
        hipEventDestroy(stop_events[i]);

        //資料由顯卡記憶體傳輸至主機記憶體
        cout << "Share Memory form Device to Host" << endl;
        hipMemcpy(r_host[i], r_device[i], blocksPerGrid * sizeof(double),
                                  hipMemcpyDeviceToHost);

        cout << "GPU "<< i <<" Elapse time for The Kernal 1 :"<< total_time[i] << " ms" << endl;
        total_time[i] = 0.0 ;
        elapsedTime[i] = 0.0 ;

        for(int j = 0; j < blocksPerGrid; j++){
            if (i == 0 && j == 0){
            r_host[0][0] = r_host[i][j];
            }
            else if (r_host[i][j] != 0){
            r_host[0][0] = r_host[0][0] + r_host[i][j];
            }
            printf("r_host[%d][%d] = %f\n", i, j, r_host[i][j]);
            printf("Ans [%d][%d] = %f\n", i, j, r_host[0][0]);
        }
        printf("r_host[0][0] = %f\n",r_host[0][0]);

        //記憶體釋放
        hipFree(r_device[i]);
        hipHostFree(r_host[i]);
        hipFree(a_device[i]);
        hipHostFree(a_host[i]);
    }
    return 0;
}
