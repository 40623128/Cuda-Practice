
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;

//const int num_GPUs = 2;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
//�ۥ[�������Ӽ�(2^30-3)
const int N               = (1 <<28);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;
//const int kernal_number = 7;
//kernel1
__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //�C�ӽu�{Ū���@�Ӥ���
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    double* a_A_host, *r_A_host;
    double* a_A_device, *r_A_device;
    float total_time = 0.0;
    //�D�����s���t
    hipHostMalloc(&a_A_host, N * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&r_A_host, blocksPerGrid * sizeof(double), hipHostMallocDefault);
    //��d���s���t
    hipMalloc(&a_A_device, N * sizeof(double));
    hipMalloc(&r_A_device, blocksPerGrid * sizeof(double));
    //�D�إͦ�
    for(int i=0;i<N;i++){
        a_A_host[i] = 1;
    }
    for(int i=0;i<blocksPerGrid;i++){
        r_A_host[i] = 0.0;
    }
    //�w�q��d�y
    hipStream_t streamA;
    //�Ыجy
    hipSetDevice(0);
    hipStreamCreate(&streamA);

    //�O����]�w(���B)
    hipMemcpyAsync(a_A_device, a_A_host, N * sizeof(double), hipMemcpyHostToDevice, streamA);
    hipMemcpyAsync(r_A_device, r_A_host, blocksPerGrid * sizeof(double), hipMemcpyHostToDevice, streamA);
    //�w�q�P�Ыض}�l�M����ƥ�(Event)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //�����}�l�ƥ�(Event)
    hipEventRecord(start, 0);
    //�B��Kernel1�i��B��

    for(int i=0;i<iters;i++){
        kernel1<<<blocksPerGrid, threadsPerBlock, 0, streamA>>>(a_A_device, r_A_device, N);
    }

    //��������ƥ�(Event)
    hipEventRecord(stop, 0);
    //���ݰ���ƥ�(Event)����
    hipEventSynchronize(stop);
    float elapsedTime;
    //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
    hipEventElapsedTime(&elapsedTime, start, stop);
    total_time = total_time + (elapsedTime / iters);
    //�ƥ󲾰�
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //�D���P�]�ƶ��O����ƻs
    hipMemcpy(r_A_host, r_A_device, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
    //varifyOutput(r_host, a_host, N);
    //����O����
    hipFree(r_A_device);
    hipFree(a_A_device);
    hipHostFree(r_A_host);
    hipHostFree(a_A_host);
    cout << "GPU Elapse time for The Kernal 1" <<" :"<< total_time << " ms" << endl;
    total_time = 0.0 ;
    return 0;
}
