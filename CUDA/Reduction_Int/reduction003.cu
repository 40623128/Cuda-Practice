
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
const int N               = (1 <<30);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;

//�ĤT��Reduction
__global__ void kernel3(float* arr, float* out, int N){
    __shared__ float s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; 
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();

    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(tid < s && i + s < N){
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;

    //�D�����s���t
    hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
    //��d���s���t
    hipMalloc(&a_device, N * sizeof(float));
    hipMalloc(&r_device, blocksPerGrid * sizeof(float));
    //�D�إͦ�
    for(int i=0;i<N;i++){
        a_host[i] = 1;
    }
    for(int i=0;i<blocksPerGrid;i++){
        r_host[i] = 0.0;
    }
    //�w�q��d�y
    hipStream_t stream;
    //�Ыجy
    hipStreamCreate(&stream);

    //�O����]�w(���B)
    hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

    //�w�q�P�Ыض}�l�M����ƥ�(Event)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //�����}�l�ƥ�(Event)
    hipEventRecord(start, 0);
    //�B��Kernel1�i��B��
    for(int i=0;i<iters;i++){
        kernel3<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
    }
    //��������ƥ�(Event)
    hipEventRecord(stop, 0);
    //���ݰ���ƥ�(Event)����
    hipEventSynchronize(stop);
    float elapsedTime;
    //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU Elapse time: " << elapsedTime / iters << " ms" << endl;
    total_time = total_time + (elapsedTime / iters);
    //�ƥ󲾰�
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //�D���P�]�ƶ��O����ƻs
    hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
    //varifyOutput(r_host, a_host, N);
    //����O����
    hipFree(r_device);
    hipFree(a_device);
    hipHostFree(r_host);
    hipHostFree(a_host);
    return 0;
}
