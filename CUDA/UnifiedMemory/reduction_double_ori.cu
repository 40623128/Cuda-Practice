
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//經測試於RTX3070最佳值為128，接著為256。
const int threadsPerBlock = 128;
//相加之元素個數(2^30-3)
const int N               = (1 <<28);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iter = 100;
//kernel1
__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    double* a_host, *r_host;
    double* a_device, *r_device;
	//主機內存分配
	hipHostMalloc(&a_host, N * sizeof(double), hipHostMallocDefault);
	hipHostMalloc(&r_host, blocksPerGrid * sizeof(double), hipHostMallocDefault);
	//顯卡內存分配
	hipMalloc(&a_device, N * sizeof(double));
	hipMalloc(&r_device, blocksPerGrid * sizeof(double));
	//題目生成
	for(int i=0;i<N;i++){
		a_host[i] = 1;
	}
	for(int i=0;i<blocksPerGrid;i++){
		r_host[i] = 0.0;
	}
	
	//定義顯卡流
	hipStream_t stream;
	//創建流
	hipStreamCreate(&stream);

	//記憶體設定(異步)
	hipMemcpyAsync(a_device, a_host, N * sizeof(double), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(double), hipMemcpyHostToDevice, stream);

	//運用Kernel1進行運算
	for(int i=0; i<iter; i++){
		kernel1<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
	}
	hipDeviceSynchronize();
	
	//主機與設備間記憶體複製
	hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
	cout << "Ans = " << r_host[0] <<" "<< endl;
	
	//釋放記憶體
	hipFree(r_device);
	hipFree(a_device);
	hipHostFree(r_host);
	hipHostFree(a_host);
	return 0;
}
