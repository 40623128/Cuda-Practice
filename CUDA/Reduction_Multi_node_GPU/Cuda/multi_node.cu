#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <mpi.h>

using namespace std;

const int threadsPerBlock = 128;
const int Total_N         = (1<<10);
const int iters           = 1;
//const int kernal_number = 7;
//kernel1


__global__ void kernel1(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //�C�ӽu�{Ū���@�Ӥ���
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}


int main(int argc, char *argv[]) {
	
	/************************
	*MPI��l��              *
	*�o���e��process �ƥ� *
	*�o���e��rank(��)  	*
	*�o��ثe�B�z�����W��	*
	************************/
	int world_size, world_rank;
	MPI_Init (&argc, &argv);
	MPI_Comm_size (MPI_COMM_WORLD, &world_size);
	MPI_Comm_rank (MPI_COMM_WORLD, &world_rank);
	char processor_name[MPI_MAX_PROCESSOR_NAME];
	int name_len;
	MPI_Get_processor_name(processor_name, &name_len);

	/*************************
	*num_gpus   ��`�IGPU�ƶq*
	*total_gpus �@�ϥ�GPU�ƶq*
	*device     �UGPU�s��    *
	*hipError_t error_code  *
	*************************/
	int num_gpus;
	int total_gpus;
	int device;
	hipError_t err;
	hipGetDeviceCount(&num_gpus);
	//num_gpus = 2;
	total_gpus = num_gpus*world_size;
	printf("node_%d_num_gpus = %d\n", world_rank,num_gpus);
	printf("total_gpus = %d\n", total_gpus);
	
	const int Node_N = Total_N/world_size;
	const int GPU_N = Node_N/num_gpus;
	const int blocksPerGrid   = (GPU_N + threadsPerBlock - 1)/threadsPerBlock;
	float total_time[total_gpus];
	double* a_host[total_gpus], *r_host[total_gpus];
	double* a_device[total_gpus], *r_device[total_gpus];
	double* All_host;
	
	//�D�إͦ�
	if(world_rank == 0){
		hipHostMalloc(&All_host, Total_N * sizeof(double));
		for(int i=0;i<Total_N;i++){
			All_host[i] = 1.0;
		}
	}
	//node�P�B
	//MPI_Barrier(MPI_COMM_WORLD);
	double *node_host = (double *)malloc(sizeof(double) * Node_N);
	//�D�ؤ��t�U�`�I
	MPI_Scatter(All_host, Node_N, MPI_DOUBLE,
				node_host, Node_N, MPI_DOUBLE,
				0, MPI_COMM_WORLD);
	printf("MPI_Scatter Finshed\n");
	for(int i = 0; i < num_gpus; i++){
		//�D�����s���t
		hipHostMalloc(&a_host[i], GPU_N * sizeof(double));
		hipHostMalloc(&r_host[i], blocksPerGrid * sizeof(double));
	}
	//printf("host Memory location Finshed\n");
	for(int i = 0; i < num_gpus; i++){
		for(int j = 0; j < GPU_N; j++){
			a_host[i][j] = node_host[i*GPU_N+j];
			//printf("a_host[%d][%d] = %f\n", i, j , a_host[i][j]);
			}
	}
	//printf("a_host location Finshed\n");
	//printf("Memory Allocation Completed\n");

	//�w�q�}�l�M����ƥ�(Event)
	hipStream_t stream[num_gpus];
	hipEvent_t start_events[num_gpus];
	hipEvent_t stop_events[num_gpus];
	float elapsedTime[num_gpus];
	
	//�w�q��d�y
	for(int i = 0; i < num_gpus; i++){
		err = hipSetDevice(i);
		err = hipGetDevice(&device);
		hipStreamCreate(&stream[i]);
		//printf("GPU %d Set Device = %d\n",i,err);
		//printf("GPU %d Get Device = %d\n",i,err);
		//printf("GPU %d Stream Define Completed\n",i);
		
		hipMalloc(&a_device[i], GPU_N * sizeof(double));
		hipMalloc(&r_device[i], blocksPerGrid * sizeof(double));



		//�O����ƻs(���B)
		hipMemcpyAsync(a_device[i], a_host[world_rank*num_gpus+i], GPU_N * sizeof(double),
						hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(r_device[i], r_host[i], blocksPerGrid * sizeof(double),
						hipMemcpyHostToDevice, stream[i]);
		//printf("Mem ERROR GPU %d = %s\n",i,hipGetErrorString(hipGetLastError()));
		//printf("Memory asynchronous Completed\n");

		//�Ыض}�l�M����ƥ�(Event)
		hipEventCreate(&start_events[i]);
		hipEventCreate(&stop_events[i]);
		//printf("Create Start & Stop Event Completed\n");

		//printf("Start Calculation\n");
		hipEventRecord(start_events[i],stream[i]);
		//�B��Kernel1�i��B��
		kernel1<<<blocksPerGrid, threadsPerBlock, 0,stream[i]>>>(a_device[i], r_device[i], GPU_N);
		//��������ƥ�(Event)
		hipEventRecord(stop_events[i],stream[i]);
		hipDeviceSynchronize();
		hipEventSynchronize(stop_events[i]);
		//printf("node_%d_GPU_%d ERROR = %s\n", world_rank, i, hipGetErrorString(hipGetLastError()));
		//printf("GPU %d Calculation Completed\n",i);

		//�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
		//printf("GPU %d Calculation time\n",i);
		hipEventElapsedTime(&elapsedTime[i], start_events[i], stop_events[i]);
		total_time[i] = total_time[i] + (elapsedTime[i] / iters);
		//printf("total_time %d = %f Calculation time\n", i, total_time[i]);
		//printf("elapsedTime %d = %f Calculation time\n", i, elapsedTime[i]);
		
		if (i ==0){
			total_time[i] = total_time[i];
		}
		else{
			total_time[i] = total_time[i-1] + total_time[i];
		}

		//printf("Event Destroy\n");
		hipEventDestroy(start_events[i]);
		hipEventDestroy(stop_events[i]);

		//��ƥ���d�O����ǿ�ܥD���O����
		//printf("Share Memory form Device to Host\n");
		hipMemcpy(r_host[i], r_device[i], blocksPerGrid * sizeof(double), hipMemcpyDeviceToHost);
		//printf("node_%d_GPU_%d Elapse time for The Kernal 1 : %f ms\n", world_rank, i, total_time[i]);
		total_time[i] = 0.0 ;
		elapsedTime[i] = 0.0 ;

		for(int j = 0; j < blocksPerGrid; j++){
			if (i == 0 && j == 0){
			r_host[0][0] = r_host[i][j];
			}
			else if (r_host[i][j] != 0){
			r_host[0][0] = r_host[0][0] + r_host[i][j];
			}
			//printf("r_host[%d][%d] = %f\n", i, j, r_host[i][j]);
			//printf("Ans [%d][%d] = %f\n", i, j, r_host[0][0]);
		}
		//printf("node_%d_GPU_%d = %f\n", world_rank, i, r_host[0][0]);
		//printf("r_host[0][0] = %f\n", r_host[0][0]);
	}
	
	double *All_Ans = (double *)malloc(sizeof(double) * world_size);;
	MPI_Gather(&r_host[0][0], 1, MPI_DOUBLE, All_Ans, 1, MPI_DOUBLE, 0,
           MPI_COMM_WORLD);
	if (world_rank == 0){
		double final_Ans;
		for(int i = 0; i < world_size; i++){
		final_Ans += All_Ans[i];
		}
		printf("final_Ans = %f\n", final_Ans);
	}
	for(int i = 0; i < num_gpus; i++){
		//�O��������
		hipFree(r_device[i]);
		hipHostFree(r_host[i]);
		hipFree(a_device[i]);
		hipHostFree(a_host[i]);
	}
	MPI_Barrier(MPI_COMM_WORLD);
	printf("node_%d_Finshed\n",world_rank);
	MPI_Finalize();
	return 0;
}


