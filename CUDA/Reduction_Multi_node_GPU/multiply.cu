/* multiply.cu */
#include<iostream>
#include<time.h>
#include <hip/hip_runtime.h>



using std::cout;using std::endl;
//const int num_node = 2;
//const int num_gpus = 8;
const int threadsPerBlock = 128;
const int N               = (1 <<25 );
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 1;

__global__ void __multiply__(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

extern "C" void launch_multiply(const int num_node, const int num_gpus)
{
    printf("num_node = %d\n"
            "num_gpus = %d\n",
           num_node,num_gpus);

    float total_time[num_gpus];
    double* a_host[num_gpus], *r_host[num_gpus];
    double* a_device[num_gpus], *r_device[num_gpus];


    //內存分配
    for(int i = 0; i < num_gpus; i++){
        //主機內存分配
        hipHostMalloc(&a_host[i], N * sizeof(double), hipHostMallocDefault);
        hipHostMalloc(&r_host[i], blocksPerGrid * sizeof(double), hipHostMallocDefault);
        //顯卡內存分配
        hipMalloc(&a_device[i], N * sizeof(double));
        hipMalloc(&r_device[i], blocksPerGrid * sizeof(double));
    }
    printf("Memory Allocation Completed\n");


    //題目生成
    printf("Generating list\n");
    for(int i = 0; i < num_gpus; i++){
        for(int j=0;j<N;j++){
            a_host[i][j] = 1.0;
            //printf("i = %d ;j = %d\n",i,j);
        }
        printf("a_host %d Generating Completed\n",i);
        for(int j=0;j<blocksPerGrid;j++){
            r_host[i][j] = 0.0;
            //printf("i = %d ;j = %d\n",i,j);
        }
        printf("r_host %d Generating Completed\n",i);
    }


    //定義顯卡流
    hipStream_t stream[num_gpus];
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipStreamCreate(&stream[i]);
    }
    printf("GPU Stream Define Completed\n");


    //記憶體設定(異步)
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipMemcpyAsync(a_device[i], a_host[i], N * sizeof(double),
                                       hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(r_device[i], r_host[i], blocksPerGrid * sizeof(double),
                                       hipMemcpyHostToDevice, stream[i]);
    }
    printf("Memory asynchronous Completed\n");

    //定義開始和停止事件(Event)
    hipEvent_t start_events[num_gpus];
    hipEvent_t stop_events[num_gpus];

    //創建開始和停止事件(Event)
    for(int i = 0; i < num_gpus; i++){
     hipSetDevice(i);
     hipEventCreate(&start_events[i]);
     hipEventCreate(&stop_events[i]);
    }
    printf("Create Start & Stop Event Completed\n");


    printf("Start Calculation\n");
    for(int j=0;j<iters;j++){
        for(int i = 0; i < num_gpus; i++){
            hipSetDevice(i);
            // In cudaEventRecord, ommit stream or set it to 0 to record
            // in the default stream. It must be the same stream as
            // where the kernel is launched.
            //紀錄開始事件(Event)
            hipEventRecord(start_events[i], stream[0]);
            //運用Kernel1進行運算
            __multiply__ <<<blocksPerGrid, threadsPerBlock, 0, stream[0]>>>(a_device[i], r_device[i], N);
            //紀錄停止事件(Event)
            hipEventRecord(stop_events[i], stream[0]);
        }
    }
    printf("Calculation Completed\n");


    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
        hipEventSynchronize(stop_events[i]);
    }
    printf("Calculation time\n");


    float elapsedTime[num_gpus];
    //計算開始事件至暫停事件所經時間
    for(int i = 0; i < num_gpus; i++){
        hipEventElapsedTime(&elapsedTime[i], start_events[i], stop_events[i]);
        total_time[i] = total_time[i] + (elapsedTime[i] / iters);
        printf("total_time %d = %f\n",i, total_time[i]);
        printf("elapsedTime %d = %f\n",i, elapsedTime[i]);
    }


    for(int i = 0; i < num_gpus; i++){
        if (i ==0){
            total_time[i] = total_time[i];
        }
        else{
        total_time[i] = total_time[i-1] + total_time[i];
        }
        }


    printf("Event Destroy\n");
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipEventDestroy(start_events[i]);
        hipEventDestroy(stop_events[i]);
    }


    printf("Share Memory form Device to Host\n");
    //資料由顯卡記憶體傳輸至主機記憶體
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipMemcpy(r_host[i], r_device[i],
                   blocksPerGrid * sizeof(double),
                   hipMemcpyDeviceToHost);
    }


    printf("Free Memory\n");
    //釋放記憶體
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipFree(r_device[i]);
        hipHostFree(r_host[i]);
        hipFree(a_device[i]);
        hipHostFree(a_host[i]);
    }


    for(int i = 0; i < num_gpus; i++){
        printf("GPU %d Elapse time for The Kernal 1 : %f ms\n",i, total_time[i]);
        total_time[i] = 0.0 ;
        elapsedTime[i] = 0.0 ;
    }
}
