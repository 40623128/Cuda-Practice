/* multiply.cu */
#include<iostream>
#include<time.h>
#include <hip/hip_runtime.h>



using std::cout;using std::endl;

const int num_gpus = 8;
const int threadsPerBlock = 128;
const int N               = (1 <<28 )/num_gpus;
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 1;

/*
__global__ void __multiply__ (const float *a, float *b)
{
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    b[i] *= a[i];
}
*/
__global__ void __multiply__(double* arr, double* out, int N){
    __shared__ double s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

extern "C" void launch_multiply(const float *a, const float *b)
{
    float total_time[num_gpus];
    double* a_host[num_gpus], *r_host[num_gpus];
    double* a_device[num_gpus], *r_device[num_gpus];


    //內存分配
    for(int i = 0; i < num_gpus; i++){
        //主機內存分配
        hipHostMalloc(&a_host[i], N * sizeof(double), hipHostMallocDefault);
        hipHostMalloc(&r_host[i], blocksPerGrid * sizeof(double), hipHostMallocDefault);
        //顯卡內存分配
        hipMalloc(&a_device[i], N * sizeof(double));
        hipMalloc(&r_device[i], blocksPerGrid * sizeof(double));
    }
    cout << "Memory Allocation Completed" << endl;



    //題目生成
    cout << "Generating list" << endl;
    for(int i = 0; i < num_gpus; i++){
        for(int j=0;j<N;j++){
            a_host[i][j] = 1.0;
            //cout <<"i =" <<i<< "; j =" <<j<< endl;
        }
        cout << "a_host "<< i <<" Generating Completed" << endl;
        for(int j=0;j<blocksPerGrid;j++){
            r_host[i][j] = 0.0;
            //cout <<"i =" <<i<< "; j =" <<j<< endl;
        }
        cout << "r_host "<< i <<" Generating Completed" << endl;
    }



    //定義顯卡流
    hipStream_t stream[num_gpus];
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipStreamCreate(&stream[i]);
    }
    cout << "GPU Stream Define Completed" << endl;



    //記憶體設定(異步)
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipMemcpyAsync(a_device[i], a_host[i], N * sizeof(double),
                                       hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(r_device[i], r_host[i], blocksPerGrid * sizeof(double),
                                       hipMemcpyHostToDevice, stream[i]);
    }
    cout << "Memory asynchronous Completed" << endl;

    //定義開始和停止事件(Event)
    hipEvent_t start_events[num_gpus];
    hipEvent_t stop_events[num_gpus];

    //創建開始和停止事件(Event)
    for(int i = 0; i < num_gpus; i++){
     hipSetDevice(i);
     hipEventCreate(&start_events[i]);
     hipEventCreate(&stop_events[i]);
    }
    cout << "Create Start & Stop Event Completed" << endl;
    cout << "Start Calculation" << endl;


    for(int j=0;j<iters;j++){
        for(int i = 0; i < num_gpus; i++){
            hipSetDevice(i);
            // In cudaEventRecord, ommit stream or set it to 0 to record
            // in the default stream. It must be the same stream as
            // where the kernel is launched.
            //紀錄開始事件(Event)
            hipEventRecord(start_events[i], stream[0]);
            //運用Kernel1進行運算
            __multiply__ <<<blocksPerGrid, threadsPerBlock, 0, stream[0]>>>(a_device[i], r_device[i], N);
            //紀錄停止事件(Event)
            hipEventRecord(stop_events[i], stream[0]);
        }
    }
    cout << "Calculation Completed" << endl;


    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
        hipEventSynchronize(stop_events[i]);
    }
    cout << "Calculation time" << endl;
    float elapsedTime[num_gpus];
    //計算開始事件至暫停事件所經時間
    for(int i = 0; i < num_gpus; i++){
        hipEventElapsedTime(&elapsedTime[i], start_events[i], stop_events[i]);
        total_time[i] = total_time[i] + (elapsedTime[i] / iters);
        cout << "total_time "<< i << "= " << total_time[i] << endl;
        cout << "elapsedTime "<< i << "= " << elapsedTime[i] << endl;
    }

    for(int i = 0; i < num_gpus; i++){
        if (i ==0){
            total_time[i] = total_time[i];
        }
        else{
        total_time[i] = total_time[i-1] + total_time[i];
        }
        }


    cout << "Event Destroy" << endl;
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipEventDestroy(start_events[i]);
        hipEventDestroy(stop_events[i]);
    }

    cout << "Share Memory form Device to Host" << endl;
    //資料由顯卡記憶體傳輸至主機記憶體
    for(int i = 0; i < num_gpus; i++){
        //創建流
        hipSetDevice(i);
        hipMemcpy(r_host[i], r_device[i],
                   blocksPerGrid * sizeof(double),
                   hipMemcpyDeviceToHost);
    }

    cout << "Free Memory" << endl;
    //釋放記憶體
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipFree(r_device[i]);
        hipHostFree(r_host[i]);
        hipFree(a_device[i]);
        hipHostFree(a_host[i]);
    }
    for(int i = 0; i < num_gpus; i++){
        cout << "GPU "<< i <<" Elapse time for The Kernal 1 :"<< total_time[i] << " ms" << endl;
        total_time[i] = 0.0 ;
        elapsedTime[i] = 0.0 ;
    }
}
