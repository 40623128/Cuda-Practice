/* multiply.cu */
#include<iostream>
#include<time.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
const int threadsPerBlock = 128;
const int iters           = 1;


/*
//Reduction 001
__global__ void __multiply__(double *arr, double *out, int N){
    __shared__ double s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
        printf("s_data[0] = %f\n",s_data[0]);
    }
}
*/


//Reduction 002
__global__ void __multiply__(double *arr, double *out, int N){
    __shared__ double s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();

    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(tid < s && i + s < N){
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
        //printf("s_data[0] = %f\n",s_data[0]);
    }
}






extern "C" double *launch_multiply(const int N ,const int num_node,
                                   double *node_host,int world_rank)
{
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    for(int i=0;i<num_gpus;i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device id: %d\n", i);
        printf("Device name: %s\n", prop.name);
    }

    const int Gpu_N = N/num_node/num_gpus;
    const int blocksPerGrid   = (Gpu_N + threadsPerBlock - 1)/threadsPerBlock;

    printf("Node %d\n"
            "N = %d\n"
            "num_node = %d\n"
            "num_gpus = %d\n"
            "node_host = %f\n",
            world_rank,N,num_node,num_gpus,node_host[0]);

    /*  內存分配
     *      主機內存分配
     */
    float total_time[num_gpus];
    double *r_host[num_gpus],*a_host[num_gpus];
    double *a_device[num_gpus], *r_device[num_gpus];
    for(int i = 0; i < num_gpus; i++){
        hipHostMalloc(&a_host[i], Gpu_N * sizeof(double), hipHostMallocDefault);
        hipHostMalloc(&r_host[i], blocksPerGrid * sizeof(double), hipHostMallocDefault);
    }
    printf("Node %d Memory Allocation Completed\n",world_rank);


    //題目生成與分配
    printf("Node %d Generating list\n",world_rank);
    for(int i = 0; i < num_gpus; i++){
        for(int j=0;j<blocksPerGrid;j++){
            r_host[i][j] = 0.0;
        }
        for(int j=0;j<Gpu_N;j++){
            a_host[i][j] = node_host[i*Gpu_N+j];
            }
    }


    //定義顯卡流
    hipStream_t stream[num_gpus];
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        //顯卡內存分配
        hipMalloc(&a_device[i], Gpu_N * sizeof(double));
        hipMalloc(&r_device[i], blocksPerGrid * sizeof(double));
        //創建流
        hipStreamCreate(&stream[i]);
    }
    printf("ERROR_hipStreamCreate = %s\n",hipGetErrorString(hipGetLastError()));
    printf("Node %d GPU Stream Define Completed\n",world_rank);

    //記憶體設定(異步)
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipMemcpyAsync(a_device[i], a_host[i], Gpu_N * sizeof(double),
                                       hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(r_device[i], r_host[i], blocksPerGrid * sizeof(double),
                                       hipMemcpyHostToDevice, stream[i]);
    }
    printf("Node %d Memory asynchronous Completed\n",world_rank);

    //定義開始和停止事件(Event)
    hipEvent_t start_events[num_gpus];
    hipEvent_t stop_events[num_gpus];


    //創建開始和停止事件(Event)
    for(int i = 0; i < num_gpus; i++){
     hipSetDevice(i);
     hipEventCreate(&start_events[i]);
     hipEventCreate(&stop_events[i]);
    }
    printf("Node %d Create Start & Stop Event Completed\n",world_rank);


    printf("Node %d Start Calculation\n",world_rank);
    for(int i = 0; i < num_gpus; i++){
        /* 設定device
        * 紀錄開始事件(Event)
        * 運用__multiply__進行運算
        * 紀錄停止事件(Event)
        */
        hipSetDevice(i);
        hipEventRecord(start_events[i], stream[i]);
        __multiply__ <<<blocksPerGrid, threadsPerBlock, 0, stream[i]>>>(a_device[i], r_device[i], Gpu_N);
        hipEventRecord(stop_events[i], stream[i]);
        hipDeviceSynchronize();
        printf("Node %d GPU %d ERROR = %s\n",world_rank,i,hipGetErrorString(hipGetLastError()));
        hipEventSynchronize(stop_events[i]);
    }
    printf("Node %d Calculation Completed\n",world_rank);

    float elapsedTime[num_gpus];
    //計算開始事件至暫停事件所經時間
    for(int i = 0; i < num_gpus; i++){
        hipEventElapsedTime(&elapsedTime[i], start_events[i], stop_events[i]);
        total_time[i] = total_time[i] + (elapsedTime[i] / iters);
        //printf("total_time %d = %f\n",i, total_time[i]);
        //printf("elapsedTime %d = %f\n",i, elapsedTime[i]);
    }


    for(int i = 0; i < num_gpus; i++){
        if (i ==0){
            total_time[i] = total_time[i];
        }
        else{
        total_time[i] = total_time[i-1] + total_time[i];
        }
        }


    printf("Node %d Event Destroy\n",world_rank);
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipEventDestroy(start_events[i]);
        hipEventDestroy(stop_events[i]);
    }

    printf("Node %d Share Memory form Device to Host\n",world_rank);
    //資料由顯卡記憶體傳輸至主機記憶體
    for(int i = 0; i < num_gpus; i++)
    {
        hipSetDevice(i);
        //cudaMemcpy(r_host[i], r_device[i],blocksPerGrid * sizeof(double),cudaMemcpyDeviceToHost);
        hipMemcpyAsync(r_host[i], r_device[i],blocksPerGrid * sizeof(double),hipMemcpyDeviceToHost,stream[i]);
        printf("ERROR_hipMemcpyAsync Node %d GPU %d ERROR = %s\n",world_rank,i,hipGetErrorString(hipGetLastError()));
        //printf("Node %d r_host[%d][0] = %f\n",world_rank,i, r_host[i][0]);
    }

    printf("Node %d Free Memory\n",world_rank);
    //釋放記憶體
    for(int i = 0; i < num_gpus; i++){
        hipSetDevice(i);
        hipFree(r_device[i]);
        //cudaFreeHost(r_host[i]);
        hipFree(a_device[i]);
        //cudaFreeHost(a_host[i]);
    }


    for(int i = 0; i < num_gpus; i++){
        printf("Node %d GPU %d Elapse time for The __multiply__ : %f ms\n",
        world_rank,i, total_time[i]);
        total_time[i] = 0.0 ;
        elapsedTime[i] = 0.0 ;
    }

    for(int i = 0; i < num_gpus; i++){
        for(int j = 0; j < blocksPerGrid; j++){
            if (i == 0 && j == 0){
            r_host[0][0] = r_host[i][j];
            }
            else if (r_host[i][j] != 0){
            r_host[0][0] = r_host[0][0] + r_host[i][j];
            }
            //printf("Node %d r_host[%d][%d] = %f\n", world_rank, i, j, r_host[i][j]);
            //printf("Node %d Ans [%d][%d] = %f\n", world_rank, i, j, r_host[0][0]);
        }
    }
    //printf("r_host[0][0] = %f\n",r_host[0][0]);

    return *r_host;
}
