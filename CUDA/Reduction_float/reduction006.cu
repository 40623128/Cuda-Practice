
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//經測試於RTX3070最佳值為128，接著為256。
const int threadsPerBlock = 128;
//相加之元素個數(2^30-3)
const int N               = (1 <<30);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;

template<unsigned int blockSize>
__device__ void warpRecude2(volatile float* s_data, int tid){ // volatile ??字很重要，保?s_data?相?的?存?元取出，?里??指gpu?存
    if(blockSize >= 64) s_data[tid] += s_data[tid + 32];   // if 是防止blockSize小于64，比如blockSize?16，那么?直接到下面
    if(blockSize >= 32) s_data[tid] += s_data[tid + 16];
    if(blockSize >= 16) s_data[tid] += s_data[tid + 8];
    if(blockSize >= 8) s_data[tid] += s_data[tid + 4];
    if(blockSize >= 4) s_data[tid] += s_data[tid + 2];
    if(blockSize >= 2) s_data[tid] += s_data[tid + 1];
}


template<unsigned int blockSize>
__global__ void reduce(float* arr, float* out, int N){
    __shared__ float s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2); // 3的第一?迭代，有一半的?程是idle的，?在把一?block的大小?小一半
    if(i < N){
        s_data[tid] = arr[i] + arr[i + blockDim.x];  // ???行原?的第一?迭代，后面代?不用?
    }else{
        s_data[tid] = 0;
    }
    __syncthreads();

    if(blockSize >= 1024){
        if(tid < 512){
            s_data[tid] += s_data[tid+512];
        }
        __syncthreads();
    }
    if(blockSize >= 512){
        if(tid < 256){
            s_data[tid] += s_data[tid+256];
        }
        __syncthreads();
    }
    if(blockSize >= 256){
        if(tid < 128){
            s_data[tid] += s_data[tid+128];
        }
        __syncthreads();
    }
    if(blockSize >= 128){
        if(tid < 64){
            s_data[tid] += s_data[tid+64];
        }
        __syncthreads();
    }

    if(tid < 32){
        warpRecude2<blockSize>(s_data, tid);
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

void kernel6(float* arr, float* out, int N, hipStream_t &stream){   // 展?所有的循?，去除循?
    switch(threadsPerBlock){
        case 1024:
            reduce<1024><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 512:
            reduce<512><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 256:
            reduce<256><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 128:
            reduce<128><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 64:
            reduce<64><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 32:
            reduce<32><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 16:
            reduce<16><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 8:
            reduce<8><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 4:
            reduce<4><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 2:
            reduce<2><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
        case 1:
            reduce<1><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(arr, out, N);break;
    }
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;

    //主機內存分配
    hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
    //顯卡內存分配
    hipMalloc(&a_device, N * sizeof(float));
    hipMalloc(&r_device, blocksPerGrid * sizeof(float));
    //題目生成
    for(int i=0;i<N;i++){
        a_host[i] = 1;
    }
    for(int i=0;i<blocksPerGrid;i++){
        r_host[i] = 0.0;
    }
    //定義顯卡流
    hipStream_t stream;
    //創建流
    hipStreamCreate(&stream);

    //記憶體設定(異步)
    hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

    //定義與創建開始和停止事件(Event)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //紀錄開始事件(Event)
    hipEventRecord(start, 0);
    //運用Kernel1進行運算
    for(int i=0;i<iters;i++){
        //kernel6<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
        kernel6(a_device, r_device, N, stream);
    }
    //紀錄停止事件(Event)
    hipEventRecord(stop, 0);
    //等待停止事件(Event)完成
    hipEventSynchronize(stop);
    float elapsedTime;
    //計算開始事件至暫停事件所經時間
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU Elapse time: " << elapsedTime / iters << " ms" << endl;
    total_time = total_time + (elapsedTime / iters);
    //事件移除
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //主機與設備間記憶體複製
    hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
    //varifyOutput(r_host, a_host, N);
    //釋放記憶體
    hipFree(r_device);
    hipFree(a_device);
    hipHostFree(r_host);
    hipHostFree(a_host);
    return 0;
}
