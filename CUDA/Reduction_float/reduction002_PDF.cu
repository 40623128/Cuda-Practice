
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//經測試於RTX3070最佳值為128，接著為256。
const int threadsPerBlock = 128;
//相加之元素個數(2^30-3)
const int N               = (1 <<30);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;
//計算平均時間之執行次數
const int times_of_average = 10;

//第一種Reduction
__global__ void kernel2(float* arr, float* out, int N) {
    __shared__ float s_data[threadsPerBlock];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    s_data[tid] = arr[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
        s_data[index] += s_data[index + s];
    }
    __syncthreads();
}

    // write result for this block to global mem
    if (tid == 0) out[blockIdx.x] = s_data[0];
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;

    for(int j=0; j<times_of_average; j++){
        //主機內存分配
        hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
        hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
        //顯卡內存分配
        hipMalloc(&a_device, N * sizeof(float));
        hipMalloc(&r_device, blocksPerGrid * sizeof(float));
        //題目生成
        for(int i=0;i<N;i++){
            a_host[i] = 1;
        }
        for(int i=0;i<blocksPerGrid;i++){
            r_host[i] = 0.0;
        }
        //定義顯卡流
        hipStream_t stream;
        //創建流
        hipStreamCreate(&stream);

        //記憶體設定(異步)
        hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

        //定義與創建開始和停止事件(Event)
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        //紀錄開始事件(Event)
        hipEventRecord(start, 0);
        //運用Kernel1進行運算
        for(int i=0;i<iters;i++){
            kernel2<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
        }
        //紀錄停止事件(Event)
        hipEventRecord(stop, 0);
        //等待停止事件(Event)完成
        hipEventSynchronize(stop);
        float elapsedTime;
        //計算開始事件至暫停事件所經時間
        hipEventElapsedTime(&elapsedTime, start, stop);
        cout << "GPU Elapse time "<<j<<" : " << elapsedTime / iters << " ms" << endl;
        total_time = total_time + (elapsedTime / iters);
        //事件移除
        hipEventDestroy(start);
        hipEventDestroy(stop);
        //主機與設備間記憶體複製
        hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
        //varifyOutput(r_host, a_host, N);
        //釋放記憶體
        hipFree(r_device);
        hipFree(a_device);
        hipHostFree(r_host);
        hipHostFree(a_host);
        //return 0;
}

cout << "GPU Elapse average time for " << times_of_average <<" times:"<< total_time/times_of_average << " ms" << endl;
return 0;
}
