
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
//�ۥ[�������Ӽ�(2^30-3)
const int N               = (1 <<28);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;
//�p�⥭���ɶ������榸��
const int times_of_average = 10;

//�Ĥ@��Reduction
__global__ void kernel1(float* arr, float* out, int N){
    __shared__ float s_data[threadsPerBlock];
    //�C�ӽu�{Ū���@�Ӥ���
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;

    for(int j=0; j<times_of_average; j++){
        //�D�����s���t
        hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
        hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
        //��d���s���t
        hipMalloc(&a_device, N * sizeof(float));
        hipMalloc(&r_device, blocksPerGrid * sizeof(float));
        //�D�إͦ�
        for(int i=0;i<N;i++){
            a_host[i] = 1;
        }
        for(int i=0;i<blocksPerGrid;i++){
            r_host[i] = 0.0;
        }
        cout << "r_host =" << sizeof(r_host[0]) <<" a_host ="<< a_host << endl;
        //�w�q��d�y
        hipStream_t stream;
        //�Ыجy
        hipStreamCreate(&stream);

        //�O����]�w(���B)
        hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

        //�w�q�P�Ыض}�l�M����ƥ�(Event)
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        //�����}�l�ƥ�(Event)
        hipEventRecord(start, 0);
        //�B��Kernel1�i��B��
        for(int i=0;i<iters;i++){
            kernel1<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
        }
        //��������ƥ�(Event)
        hipEventRecord(stop, 0);
        //���ݰ���ƥ�(Event)����
        hipEventSynchronize(stop);
        float elapsedTime;
        //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
        hipEventElapsedTime(&elapsedTime, start, stop);
        cout << "GPU Elapse time "<<j<<" : " << elapsedTime / iters << " ms" << endl;
        total_time = total_time + (elapsedTime / iters);
        //�ƥ󲾰�
        hipEventDestroy(start);
        hipEventDestroy(stop);
        //�D���P�]�ƶ��O����ƻs
        hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
        //varifyOutput(r_host, a_host, N);
        cout << "r_host =" << r_host <<" a_host ="<< a_host << endl;
        //����O����
        hipFree(r_device);
        hipFree(a_device);
        hipHostFree(r_host);
        hipHostFree(a_host);
        //return 0;
}

cout << "GPU Elapse average time for " << times_of_average <<" times:"<< total_time/times_of_average << " ms" << endl;
return 0;
}
