
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//�g���թ�RTX3070�̨έȬ�128�A���۬�256�C
const int threadsPerBlock = 128;
const int N               = (1 <<30);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;

//�ĤG��Reduction
__global__ void kernel2(float* arr, float* out, int N){
    __shared__ float s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; 
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    // s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        int index = tid * 2 * s;
        if((index + s) < blockDim.x && (blockIdx.x * blockDim.x + index + s) < N){
            s_data[index] += s_data[index + s];
        }
        __syncthreads();
    }

    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;
    //�D�����s���t
    hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
    //��d���s���t
    hipMalloc(&a_device, N * sizeof(float));
    hipMalloc(&r_device, blocksPerGrid * sizeof(float));
    //�D�إͦ�
    for(int i=0;i<N;i++){
        a_host[i] = 1;
    }
    for(int i=0;i<blocksPerGrid;i++){
        r_host[i] = 0.0;
    }
    //�w�q��d�y
    hipStream_t stream;
    //�Ыجy
    hipStreamCreate(&stream);

    //�O����]�w(���B)
    hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

    //�w�q�P�Ыض}�l�M����ƥ�(Event)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //�����}�l�ƥ�(Event)
    hipEventRecord(start, 0);
    //�B��Kernel1�i��B��
    for(int i=0;i<iters;i++){
        kernel2<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
    }
    //��������ƥ�(Event)
    hipEventRecord(stop, 0);
    //���ݰ���ƥ�(Event)����
    hipEventSynchronize(stop);
    float elapsedTime;
    //�p��}�l�ƥ�ܼȰ��ƥ�Ҹg�ɶ�
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU Elapse time: " << elapsedTime / iters << " ms" << endl;
    total_time = total_time + (elapsedTime / iters);
    //�ƥ󲾰�
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //�D���P�]�ƶ��O����ƻs
    hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
    //varifyOutput(r_host, a_host, N);
    //����O����
    hipFree(r_device);
    hipFree(a_device);
    hipHostFree(r_host);
    hipHostFree(a_host);
    return 0;
}
