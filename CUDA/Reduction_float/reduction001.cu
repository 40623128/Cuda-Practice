
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
using namespace std;
//經測試於RTX3070最佳值為128，接著為256。
const int threadsPerBlock = 128;
//相加之元素個數(2^30-3)
const int N               = (1 <<30);
const int blocksPerGrid   = (N + threadsPerBlock - 1)/threadsPerBlock;
const int iters           = 100;

//第一種Reduction
__global__ void kernel1(float* arr, float* out, int N){
    __shared__ float s_data[threadsPerBlock];
    //每個線程讀取一個元素
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < N){
        s_data[tid] = arr[i];
    }
    __syncthreads();
    //s = 1 2 4 8 16 32 64
    for(int s = 1; s < blockDim.x; s*=2){
        if(tid % (2*s) == 0 && i + s <N){
            s_data[tid] +=
            s_data[tid + s];
        }
        __syncthreads();
    }
    if(tid == 0){
        out[blockIdx.x] = s_data[0];
    }
}

int main(){
    float* a_host, *r_host;
    float* a_device, *r_device;
    float total_time = 0.0;

    //主機內存分配
    hipHostMalloc(&a_host, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&r_host, blocksPerGrid * sizeof(float), hipHostMallocDefault);
    //顯卡內存分配
    hipMalloc(&a_device, N * sizeof(float));
    hipMalloc(&r_device, blocksPerGrid * sizeof(float));
    //題目生成
    for(int i=0;i<N;i++){
        a_host[i] = 1;
    }
    for(int i=0;i<blocksPerGrid;i++){
        r_host[i] = 0.0;
    }
    //定義顯卡流
    hipStream_t stream;
    //創建流
    hipStreamCreate(&stream);

    //記憶體設定(異步)
    hipMemcpyAsync(a_device, a_host, N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(r_device, r_host, blocksPerGrid * sizeof(float), hipMemcpyHostToDevice, stream);

    //定義與創建開始和停止事件(Event)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //紀錄開始事件(Event)
    hipEventRecord(start, 0);
    //運用Kernel1進行運算
    for(int i=0;i<iters;i++){
        kernel1<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a_device, r_device, N);
    }
    //紀錄停止事件(Event)
    hipEventRecord(stop, 0);
    //等待停止事件(Event)完成
    hipEventSynchronize(stop);
    float elapsedTime;
    //計算開始事件至暫停事件所經時間
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU Elapse time: " << elapsedTime / iters << " ms" << endl;
    total_time = total_time + (elapsedTime / iters);
    //事件移除
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //主機與設備間記憶體複製
    hipMemcpy(r_host, r_device, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
    //varifyOutput(r_host, a_host, N);
    //釋放記憶體
    hipFree(r_device);
    hipFree(a_device);
    hipHostFree(r_host);
    hipHostFree(a_host);
    return 0;
}
